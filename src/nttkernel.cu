#include "hip/hip_runtime.h"
#include <cmath>		/* log2(), pow() */
#include <cstdint>		/* uint64_t */
#include <cstdlib> 		/* malloc() */
#include <iostream>
#include <hip/hip_runtime.h>

#include "../include/utils.h"	/* bit_reverse(), modExp(), modulo() */
#include "../include/utils.cuh"



__global__
void inPlaceNTT_kernel(uint64_t p, uint64_t n, uint64_t r, uint64_t *d_vec, uint64_t *d_result) {
	uint64_t i = 1;//blockIdx.x + 1; 						// this will be given to thread according to block index

	uint64_t m = powf(2.0, (float)i);					// calculated within kernel
	uint64_t k_ = (p - 1)/m;							// we need to pass p into kernel
	uint64_t a = modExp_k(r, k_, p);					// calculated within kernel

	int s = blockIdx.x*blockDim.x + threadIdx.x; 
    int t = blockDim.x*gridDim.x;

	for(uint64_t j = 0; j < n; j += m ) { 
		d_result[j] += 1;
	}

	// for(uint64_t j = s; j < n; j += (m * stride)) {
	// 	for(uint64_t k = 0; k < m/2; k++) { 
	// 		uint64_t j1 = j+k;
	// 		uint64_t j2 = j1 + (m/2);

	// 		if(j1 < n && j2 < n) {
	// 			uint64_t factor1 = d_result[j1];
	// 			uint64_t factor2 = modulo_k(modExp_k(a, k, p) * d_result[j2], p);
		
	// 			d_result[j1] 		= modulo_k(factor1 + factor2, p);
	// 			d_result[j2] 		= modulo_k(factor1 - factor2, p);
	// 		}
	// 	}
	// }
	

}

/**
 * Perform an in-place iterative breadth-first decimation-in-time Cooley-Tukey NTT on an input vector and return the result
 *
 * @param vec 	The input vector to be transformed
 * @param n	The size of the input vector
 * @param p	The prime to be used as the modulus of the transformation
 * @param r	The primitive root of the prime
 * @param rev	Whether to perform bit reversal on the input vector
 * @return 	The transformed vector
 */
__host__
uint64_t *inPlaceNTT_DIT_parallel(uint64_t *h_vec, uint64_t n, uint64_t p, uint64_t r, bool rev){
	uint64_t *h_result;
	hipError_t err;

	h_result = (uint64_t *) malloc(n*sizeof(uint64_t));

	if(rev) {
		h_result = bit_reverse(h_vec, n);
	} else {
		for(uint64_t i = 0; i < n; i++) {
			h_result[i] = h_vec[i];
		}
	}

	/* kernel stuff */ 
	// bulk of parallelization
	dim3 dim_grid(1, 1, 1);		// blocks for each of the first for loop iterations
	dim3 dim_block(12, 1, 1);		// starting with one thread per block

	//int block_size = 128;
	//int num_blocks = (n + block_size - 1) / block_size;

	// setting up vector for kernel
	uint64_t size = n * sizeof(uint64_t);
    uint64_t *d_vec;
	err = hipMalloc((void **) &d_vec, size);
	if(err != hipSuccess) { 
		std::cout << "cuda error: something went wrong allocating on device\n";
	}

	err = hipMemcpy(d_vec, h_vec, size, hipMemcpyHostToDevice);
	if(err != hipSuccess) { 
		std::cout << "cuda error: something went wrong copying to device\n";
	}

    // setting up results vector
    uint64_t *d_result;
    hipMalloc((void **) &d_result, size);
    hipMemcpy(d_result, h_result, size, hipMemcpyHostToDevice);


	inPlaceNTT_kernel<<<dim_grid, dim_block>>>(p, n, r, d_vec, d_result);


	err = hipDeviceSynchronize();
	if(err != hipSuccess) { 
		std::cout << "cuda error: something went wrong with threads\n";
	}

	hipMemcpy(h_result, d_result, size, hipMemcpyDeviceToHost);
	hipFree(d_vec);
	hipFree(d_result);
	/* end of kernel stuff */
	

	return h_result;
}


